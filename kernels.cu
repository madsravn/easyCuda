#include "hip/hip_runtime.h"
#include "kernels.h"
#include "helpers.h"
#include <iostream>
#include <cmath>


__global__
void blur(unsigned char* input_image, unsigned char* output_image, int width, int height) {

    const unsigned int offset = blockIdx.x*blockDim.x + threadIdx.x;

    int x = offset % width;
    int y = (offset-x)/width;

    int fsize = 5; // Filter size

    if(offset < width*height) {

        float output_red = 0;
        float output_green = 0;
        float output_blue = 0;

        int hits = 0;
        for(int ox = -fsize; ox < fsize+1; ++ox) {
            for(int oy = -fsize; oy < fsize+1; ++oy) {
                if((x+ox) > -1 && (x+ox) < width && (y+oy) > -1 && (y+oy) < height) {
                    const int currentoffset = (offset+ox+oy*width)*3;
                    output_red += input_image[currentoffset]; 
                    output_green += input_image[currentoffset+1];
                    output_blue += input_image[currentoffset+2];
                    hits++;
                }
            }
        }
        output_image[offset*3] = output_red/hits;
        output_image[offset*3+1] = output_green/hits;
        output_image[offset*3+2] = output_blue/hits;
    }
}

__global__
void detect_yellow(unsigned char* input_image, unsigned char* output_image, int width, int height) {

    const unsigned int offset = blockIdx.x + threadIdx.x * blockDim.x;

    int yellow_or_not_pixel;

    //verify if the pixel is yellow or around
    if(
    input_image[offset*3] >= 180 &&
    input_image[offset*3+1] >= 180 &&
    input_image[offset*3+2] <= 155
    ){
        yellow_or_not_pixel = 255;
    } else {
        yellow_or_not_pixel = 0;
    }

    output_image[offset*3] = yellow_or_not_pixel;
    output_image[offset*3+1] = yellow_or_not_pixel;
    output_image[offset*3+2] = yellow_or_not_pixel;

}


__global__
void negative(unsigned char* input_image, unsigned char* output_image, int width, int height) {

    const unsigned int offset = blockIdx.x + threadIdx.x * blockDim.x;
    
    output_image[offset*3] = 255 - input_image[offset*3];
    output_image[offset*3+1] = 255 - input_image[offset*3+1];
    output_image[offset*3+2] = 255 - input_image[offset*3+2];
    
}


__global__
void grayscale(unsigned char* input_image, unsigned char* output_image, int width, int height) {

    const unsigned int offset = blockIdx.x + threadIdx.x * blockDim.x;
    
    //origin: image processing classes && https://www.baeldung.com/cs/convert-rgb-to-grayscale

    int gray = (input_image[offset*3] * 0.3) + (input_image[offset*3+1] * 0.59) + (input_image[offset*3+2] * 0.11);
    output_image[offset*3] = gray;
    output_image[offset*3+1] = gray;
    output_image[offset*3+2] = gray;

}


__global__
void sepia(unsigned char* input_image, unsigned char* output_image, int width, int height) {

    const unsigned int offset = blockIdx.x + threadIdx.x * blockDim.x;
    
    //origin: https://www.geeksforgeeks.org/image-processing-in-java-colored-image-to-sepia-image-conversion/

    int newRed = (input_image[offset*3] * 0.393) + (input_image[offset*3+1] * 0.769) + (input_image[offset*3+2] * 0.189);
    int newGreen = (input_image[offset*3] * 0.349) + (input_image[offset*3+1] * 0.686) + (input_image[offset*3+2] * 0.168);
    int newBlue = (input_image[offset*3] * 0.272) + (input_image[offset*3+1] * 0.534) + (input_image[offset*3+2] * 0.131);
    
    newRed > 255 ? newRed = 255 : newRed = newRed;
    newGreen > 255 ? newGreen = 255 : newGreen = newGreen;
    newBlue > 255 ? newBlue = 255 : newBlue = newBlue;

    output_image[offset*3] = newRed;
    output_image[offset*3+1] = newGreen;
    output_image[offset*3+2] = newBlue;

}


__global__
void black_and_white(unsigned char* input_image, unsigned char* output_image, int width, int height) {

    const unsigned int offset = blockIdx.x + threadIdx.x * blockDim.x;

    int black_or_white_pixel;

    int average_rgb = (input_image[offset*3] + input_image[offset*3+1]  + input_image[offset*3+2] * 0.11) / 3;

    average_rgb >= 100 ? black_or_white_pixel = 255 : black_or_white_pixel = 0;

    output_image[offset*3] = black_or_white_pixel;
    output_image[offset*3+1] = black_or_white_pixel;
    output_image[offset*3+2] = black_or_white_pixel;

}


void filter (unsigned char* input_image, unsigned char* output_image, int width, int height, int filter_id) {

    unsigned char* dev_input;
    unsigned char* dev_output;
    getError(hipMalloc( (void**) &dev_input, width*height*3*sizeof(unsigned char)));
    getError(hipMemcpy( dev_input, input_image, width*height*3*sizeof(unsigned char), hipMemcpyHostToDevice ));
 
    getError(hipMalloc( (void**) &dev_output, width*height*3*sizeof(unsigned char)));

    dim3 blockDims(512,1,1);
    dim3 gridDims((unsigned int) ceil((double)(width*height*3/blockDims.x)), 1, 1 );

    switch (filter_id){
        case 0:
            blur<<<gridDims, blockDims>>>(dev_input, dev_output, width, height);
            break;
        case 1:
            black_and_white<<<gridDims, blockDims>>>(dev_input, dev_output, width, height);
            break;
        case 2:
            grayscale<<<gridDims, blockDims>>>(dev_input, dev_output, width, height);
            break;
        case 3:
            negative<<<gridDims, blockDims>>>(dev_input, dev_output, width, height);
            break;
        case 4:
            sepia<<<gridDims, blockDims>>>(dev_input, dev_output, width, height);
            break;
        case 5:
            detect_yellow<<<gridDims, blockDims>>>(dev_input, dev_output, width, height);
            break;
    }

    getError(hipMemcpy(output_image, dev_output, width*height*3*sizeof(unsigned char), hipMemcpyDeviceToHost ));

    getError(hipFree(dev_input));
    getError(hipFree(dev_output));

}

